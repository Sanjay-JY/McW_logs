
#include <hip/hip_runtime.h>
#include <chrono>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>
const int I_WIDTH = 1024;
const int O_WIDTH = 1018; // 1024-2*r

__global__ void convolution_1d(int *array, int *mask, int *result, int n, int m) {

  int tid = blockIdx.x * O_WIDTH + threadIdx.x;
  int r = m / 2;
  int start = tid - r;
  __shared__ int s_array[I_WIDTH];
  int temp = 0;
  if((start>=0)&&start<n)
  {
    s_array[threadIdx.x]=array[start];
  }
  else
  {
    s_array[threadIdx.x]=0;
  }
  __syncthreads();
  if(threadIdx.x<O_WIDTH)
  {
    temp=0;
    for(int i=0;i<m;i++)
    {
        temp+=mask[i]*s_array[i+threadIdx.x];
    }
    result[tid]=temp;
    __syncthreads();
  }
}

void verify_result(int *array, int *mask, int *result, int n, int m) {

  int radius = m / 2;
  int temp;
  int start;
  for (int i = 0; i < n; i++) {
    start = i - radius;
    temp = 0;
    for (int j = 0; j < m; j++) {
      if ((start + j >= 0) && (start + j < n)) {
        temp += array[start + j] * mask[j];
      }
    }
    assert(temp == result[i]);
  }

}

int main() {

  int n = 1<<18;   //1<<19 doesn't work
  std::cout<<n<<"\n";
  int bytes_n = n * sizeof(int);
  int m = 7;
  int bytes_m = m * sizeof(int);

  
  std::vector<int> h_array(n);
  for(int i=0;i<n;i++)
  {
    h_array[i]=i;
  }

  std::vector<int> h_mask(m);
  for(int i=0;i<m;i++)
  {
    h_mask[i]=i;
  }
  
  std::vector<int> h_result(n);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  int *d_array, *d_mask, *d_result;
  hipMalloc(&d_array, bytes_n);
  hipMalloc(&d_mask, bytes_m);
  hipMalloc(&d_result, bytes_n);

  hipMemcpy(d_array, h_array.data(), bytes_n, hipMemcpyHostToDevice);
  hipMemcpy(d_mask, h_mask.data(), bytes_m, hipMemcpyHostToDevice);

  int THREADS = I_WIDTH;
  int GRID = n/O_WIDTH+1;     

  convolution_1d<<<GRID, THREADS>>>(d_array, d_mask, d_result, n, m);

  hipError_t hipError_t = hipGetLastError();
  if (hipError_t != hipSuccess) {
      std::cerr << "Kernel launch failed: " << hipGetErrorString(hipError_t) << std::endl;
  }

  hipMemcpy(h_result.data(), d_result, bytes_n, hipMemcpyDeviceToHost);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "GPU : "<<milliseconds << "\n";

  verify_result(h_array.data(), h_mask.data(), h_result.data(), n, m);

  std::cout << "COMPLETED SUCCESSFULLY\n";

  hipFree(d_result);
  hipFree(d_mask);
  hipFree(d_array);

  return 0;
}


