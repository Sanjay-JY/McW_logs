#include "hip/hip_runtime.h"
// Copyright (C) 2023 Associated Universities, Inc. Washington DC, USA.
// This stand alone module is an adaptation of a unit test of a specific kernel in the
// Cyclid package (a package for Cyclic Spectroscopy processing)
// To build using CUDA 11.6:
// nvcc -c cyclid_gpu.cu -o cyclid_gpu.o -dc
// nvcc -o cyclid_gpu cyclid_gpu.o

#include <stdio.h>
#include <complex>
#include <assert.h>
#include <time.h>
#include<hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

// part of a larger structure that plays a larger
// role in the full pipeline
struct cycfold_struct {
    unsigned ncyc;
    unsigned nlag;
    unsigned nchanPfb;
    size_t numPhaseBins;
    unsigned numTimeSamplesHfft;
    unsigned nBlocks;
};

// TBF: constant for this gpu?  Maybe query?
int GPU_BLOCK_SIZE = 256*4;
const int BLOCK_SIZE=256;

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
    #if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    #endif
    return result;
}

__global__ void lookuptable_kernel(float2 *in1, float2 *in2, int2 *d_lookup, float2 *d_xxresult, float2 *d_yyresult, float2 *d_xyresult, float2 *d_yxresult, float2 *d_xxtemp, float2 *d_yytemp,float2 *d_xytemp,float2 *d_yxtemp) {
    
    int rowIdx = (blockIdx.x * blockDim.x + threadIdx.x)/((1003/BLOCK_SIZE)+1);
    int threadrow = (blockIdx.x * blockDim.x + threadIdx.x)%((1003/BLOCK_SIZE)+1);
    // Check if the thread index is within the valid range (less than or equal to 16640)

    //cooperative_groups::grid_group g = cooperative_groups::this_grid(); 
    

    if (rowIdx < 16640) {
        float2 sumxx,sumyy,sumxy,sumyx;
        sumxx.x = 0.0f;
        sumxx.y = 0.0f;
        sumyy.x = 0.0f;
        sumyy.y = 0.0f;
        sumxy.x = 0.0f;
        sumxy.y = 0.0f;
        sumyx.x = 0.0f;
        sumyx.y = 0.0f;

        int start=(threadrow*BLOCK_SIZE)+1;
        int end=start+BLOCK_SIZE-1;

        if(end>=1003) end=1002;
        
        //d_yytemp[rowIdx].y=rowIdx;

        if(rowIdx==0)
        {
            d_xytemp[blockIdx.x * blockDim.x + threadIdx.x].x=start;
            d_xytemp[blockIdx.x * blockDim.x + threadIdx.x].y=end;
        }

        for (int colIdx = start; colIdx <= end; colIdx++) {

            int2 current = d_lookup[rowIdx * 1003 + colIdx];


            // if(rowIdx==0)
            // {
            //     d_yytemp[colIdx].x=colIdx;
            //     //d_xxtemp[blockIdx.x * blockDim.x + threadIdx.x].y=end;
            // }

            //d_xxtemp[rowIdx]=threadrow * 256 + colIdx;
            if(current.x==-1) break;

            int current_x=current.x;
            int current_y=current.y;

            float in1y_x=in1[current_y].x;
            float in1x_x=in1[current_x].x;
            float in1y_y=in1[current_y].y;
            float in1x_y=in1[current_x].y;

            float in2y_x=in2[current_y].x;
            float in2x_x=in2[current_x].x;
            float in2y_y=in2[current_y].y;
            float in2x_y=in2[current_x].y;

            //if(rowIdx==0)
            // {
            //     d_yytemp[colIdx].x=colIdx;
            //     //d_xxtemp[blockIdx.x * blockDim.x + threadIdx.x].y=end;
            // }

            float2 product;

            //XX Corelation
            product.x = (in1y_x * in1x_x) - (in1y_y *-1.0* in1x_y);
            product.y = (in1y_x * -1.0*in1x_y) + (in1y_y * in1x_x);
            product.y = -1.0*product.y;
            //atomicAdd(&d_xxresult[rowIdx].x, product.x);
            // if(rowIdx==0){
            //     printf("Sum Before: %f\n",sumxx.x);
            // }
            sumxx.x += product.x;
            // if(rowIdx==0){
            //     printf("Sum After: %f\t Pro:%f\n",sumxx.x,product.x);
            // }
            sumxx.y += product.y;
            // if(rowIdx==2)
            //     printf("%f\n",product.x);
            
            if(rowIdx==0)
            {
                d_yytemp[colIdx].x=product.x;
                //d_xxtemp[blockIdx.x * blockDim.x + threadIdx.x].y=end;
            }

            //YY Corelation
            product.x = (in2y_x * in2x_x) - (in2y_y *-1.0* in2x_y);
            product.y = (in2y_x * -1.0*in2x_y) + (in2y_y * in2x_x);
            product.y = -1.0*product.y;
            sumyy.x += product.x;
            sumyy.y += product.y;

            //XY Corelation
            product.x = (in1y_x * in2x_x) - (in1y_y *-1.0* in2x_y);
            product.y = (in1y_x * -1.0*in2x_y) + (in1y_y * in2x_x);
            product.y = -1.0*product.y;
            sumxy.x += product.x;
            sumxy.y += product.y;

            //YX Corelation
            product.x = (in2y_x * in1x_x) - (in2y_y *-1.0* in1x_y);
            product.y = (in2y_x * -1.0*in1x_y) + (in2y_y * in1x_x);
            product.y = -1.0*product.y;
            sumyx.x += product.x;
            sumyx.y += product.y;
        }


        
        if(rowIdx==0)
        {
            //printf("sumxx.x->%f\n",sumxx.x);
            d_yytemp[threadrow].y=sumxx.x;
            //d_yytemp[threadrow].y=sumxx.y;
        }
        

        //g.sync();

        float final;
        final=0;
        if(threadrow==0)
        {
            for(int i=0;i<4;i++)
            {
                final+=d_yytemp[i].y;
            }
        }
        //printf("%d\n",rowIdx);
        //d_xxresult[rowIdx].x = final;
        atomicAdd(&d_xxresult[rowIdx].x, sumxx.x);
        atomicAdd(&d_xxresult[rowIdx].y, sumxx.y);
        atomicAdd(&d_yyresult[rowIdx].x, sumyy.x);
        atomicAdd(&d_yyresult[rowIdx].y, sumyy.y);
        atomicAdd(&d_xyresult[rowIdx].x, sumxy.x);
        atomicAdd(&d_xyresult[rowIdx].y, sumxy.y);
        atomicAdd(&d_yxresult[rowIdx].x, sumyx.x);
        atomicAdd(&d_yxresult[rowIdx].y, sumyx.y);
        
        //atomicAdd(&outXX[outIdx].x, tmp.x);
        // d_xxtemp[blockIdx.x].x=sumxx.x;
        // d_xxtemp[blockIdx.x].y=sumxx.y;
        // d_yytemp[blockIdx.x].x=sumyy.x;
        // d_yytemp[blockIdx.x].y=sumyy.y;
        // d_xytemp[blockIdx.x].x=sumxy.x;
        // d_xytemp[blockIdx.x].y=sumxy.y;
        // d_yxtemp[blockIdx.x].x=sumyx.x;
        // d_yxtemp[blockIdx.x].y=sumyx.y;

        // g.sync();

        // if(threadIdx.x%3==0)
        // {
        //     sumxx.x=0;
        //     sumxx.y=0;
        //     sumyy.x=0;
        //     sumyy.y=0;
        //     sumxy.x=0;
        //     sumxy.y=0;
        //     sumyx.x=0;
        //     sumyx.y=0;
        //     for(int i=0;i<3;i++)
        //     {
        //         sumxx.x+=d_xxtemp[i].x;
        //         sumxx.y+=d_xxtemp[i].y;
        //         sumyy.x+=d_yytemp[i].x;
        //         sumyy.y+=d_yytemp[i].y;
        //         sumxy.x+=d_xytemp[i].x;
        //         sumxy.y+=d_xytemp[i].y;
        //         sumyx.x+=d_yxtemp[i].x;
        //         sumyx.y+=d_yxtemp[i].y;
        //     }
        // }
        // d_xxresult[rowIdx].x = sumxx.x;
        // d_xxresult[rowIdx].y = sumxx.y;
        // d_yyresult[rowIdx].x = sumyy.x;
        // d_yyresult[rowIdx].y = sumyy.y;
        // d_xyresult[rowIdx].x = sumxy.x;
        // d_xyresult[rowIdx].y = sumxy.y;
        // d_yxresult[rowIdx].x = sumyx.x;
        // d_yxresult[rowIdx].y = sumyx.y;

    }
}

void complexConjMult(float2 in1, float2 in2, float2 *tmp) {
    tmp->x = (in2.x * in1.x) - (in2.y * -1.0 * in1.y);
    tmp->y = (in2.x * -1.0 * in1.y) + (in2.y * in1.x);
    tmp->y = -1.0 * tmp->y;
}


// woarker function for tests of the corr_accum kernel
int test_cyclid_corr_accum(cycfold_struct *cs, unsigned *phaseBins, bool maxOccupancy, bool time) {
    printf("test_cyclid_corr_accum\n");
    fflush(stdout);

    bool verbose = false;
    int nlag = cs->nlag; //(ncyc/2) + 1;
    size_t inSize = cs->numTimeSamplesHfft; //8;
    size_t inSize2 = inSize - nlag - 1;
    int phaseBinLookupSize = (2*inSize) + nlag - 2;
    int nPhaseBins = cs->numPhaseBins;
    int nchan = cs->nchanPfb;
    int ichan = 0;
    int iblock = 0;

    float2 *in, *iny;
    in = (float2 *)malloc(inSize * sizeof(float2));
    iny = (float2 *)malloc(inSize * sizeof(float2));
    memset(in, 0, inSize*sizeof(float2));
    memset(iny, 0, inSize*sizeof(float2));

    int maxValue = 127; //255 causes overflow problems?;
    int value = 0;

    // setting the fractional parts to anything but 0
    // keeps these tests from failing!
    // TBF: I think this is the fact that atomicAdd is not
    // reproducible - floating point errors are not associative?
    float fvalue = 0.5;
    float imgDiv = 2.0;
    for (int i = 0; i<inSize; i++) {
        in[i].x = ((float)value) + fvalue;
        in[i].y = ( (float)((float)value)/imgDiv) + fvalue;
        // unimaginative population of the second polarization
        iny[i].x = in[i].x;
        iny[i].y = in[i].y;
        value++;
        if (value + fvalue>maxValue)
            value=0;
    }
    
    // compute expected results
    int phaseBinIdx, phaseBin, expIdx;
    float2 tmp, in1, in2;
    float2 *exp;
    size_t profileSize = nPhaseBins * nchan * nlag;
    exp = (float2 *)malloc(profileSize * sizeof(float2));
    memset(exp, 0, profileSize*sizeof(float2));

    //Reference Code
    for (int i = 0; i<inSize2; i++) {
        in1 = in[i];
        for (int ilag=0; ilag<nlag; ilag++) {
            in2 = in[i + ilag];
            complexConjMult(in1, in2, &tmp);
            phaseBinIdx = (2*i)+ilag;
            phaseBin = phaseBins[phaseBinIdx];
            expIdx = (phaseBin * nlag * nchan) + (nlag * ichan) + ilag;
            exp[expIdx].x += tmp.x;
            exp[expIdx].y += tmp.y;
        }
    }

    //Lookup Table Creation
    int2 *lookuptable;
    lookuptable= (int2 *)malloc(16640*1003* sizeof(int2));
    fflush(stdout);
    for(int c=0;c<16640*1003;c++)
    {
        lookuptable[c].x=-1;
    }

    //clock_t start_time = clock();
    for (int i = 0; i<inSize2; i++) {
        for (int ilag=0; ilag<nlag; ilag++) {
            phaseBinIdx = (2*i)+ilag;
            phaseBin = phaseBins[phaseBinIdx];
            expIdx = (phaseBin * nlag * nchan) + (nlag * ichan) + ilag;
            int index =(lookuptable[expIdx*1003].x);
            if(index==-1)
            {
                lookuptable[expIdx*1003].x=1;
                lookuptable[expIdx*1003 + (lookuptable[expIdx*1003].x)].x = i;
                lookuptable[expIdx*1003 + (lookuptable[expIdx*1003].x)].y = i+ilag;
                lookuptable[expIdx*1003].x=(lookuptable[expIdx*1003].x)+1;
            }
            else
            {
                lookuptable[expIdx*1003 +(lookuptable[expIdx*1003].x)].x=i;
                lookuptable[expIdx*1003 + (lookuptable[expIdx*1003].x)].y=i+ilag;
                lookuptable[expIdx*1003].x= (lookuptable[expIdx*1003].x)+1;
            }
        }
    }
    // clock_t end_time = clock();
    // double elapsed_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
    // printf("Elapsed time: %.6f seconds\n", elapsed_time);
    
    int2 *d_lookup;
    float2 *d_xxresult,*d_yyresult,*d_xyresult,*d_yxresult,*resultxx,*resultyy,*resultxy,*resultyx;
    float2 *tempxx,*tempyy,*tempxy,*tempyx,*d_xxtemp,*d_yytemp,*d_xytemp,*d_yxtemp;

    int temp_size=16640;
    tempxx=(float2 *)malloc(temp_size*sizeof(float2));
    tempyy=(float2 *)malloc(temp_size*sizeof(float2));
    tempxy=(float2 *)malloc(temp_size*sizeof(float2));
    tempyx=(float2 *)malloc(temp_size*sizeof(float2));
    for(int i=0;i<temp_size;i++)
    {
        tempxx[i].x=0;
        tempxx[i].y=0;
        tempyy[i].x=0;
        tempyy[i].y=0;
        tempxy[i].x=0;
        tempxy[i].y=0;
        tempyx[i].x=0;
        tempyx[i].y=0;
    }

    resultxx= (float2 *)malloc(16640*sizeof(float2));
    resultyy= (float2 *)malloc(16640*sizeof(float2));
    resultxy= (float2 *)malloc(16640*sizeof(float2));
    resultyx= (float2 *)malloc(16640*sizeof(float2));
    printf("Size: %d\n",sizeof(float2));
    for(int i=0;i<16640;i++)
    {
        resultxx[i].x=0.0f;
        resultxx[i].y=0.0f;
        resultyy[i].x=0.0f;
        resultyy[i].y=0.0f;
        resultxy[i].x=0.0f;
        resultxy[i].y=0.0f;
        resultyx[i].x=0.0f;
        resultyx[i].y=0.0f;
    }

    hipMalloc((void**)&d_lookup,16640*1003 * sizeof(int2));
    hipMalloc((void**)&d_xxresult, 16640* sizeof(float2));
    hipMalloc((void**)&d_yyresult, 16640* sizeof(float2));
    hipMalloc((void**)&d_xyresult, 16640* sizeof(float2));
    hipMalloc((void**)&d_yxresult, 16640* sizeof(float2));

    hipMalloc((void**)&d_xxtemp, temp_size* sizeof(float2));
    hipMalloc((void**)&d_yytemp, temp_size* sizeof(float2));
    hipMalloc((void**)&d_xytemp, temp_size* sizeof(float2));
    hipMalloc((void**)&d_yxtemp, temp_size* sizeof(float2));
    
    hipMemcpy(d_lookup,lookuptable, 16640*1003 * sizeof(int2), hipMemcpyHostToDevice);
    hipMemcpy(d_xxresult,resultxx, 16640* sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(d_yyresult,resultyy, 16640* sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(d_xyresult,resultxy, 16640* sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(d_yxresult,resultyx, 16640* sizeof(float2), hipMemcpyHostToDevice);

    hipMemcpy(d_xxtemp,tempxx, temp_size* sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(d_yytemp,tempyy, temp_size* sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(d_xytemp,tempxy, temp_size* sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(d_yxtemp,tempyx, temp_size* sizeof(float2), hipMemcpyHostToDevice);

    float2 *in_gpu, *iny_gpu;
    unsigned *phaseBins_gpu;
    hipMalloc((float2 **)&in_gpu, inSize*sizeof(float2));
    hipMalloc((float2 **)&iny_gpu, inSize*sizeof(float2));
    hipMemcpy(in_gpu, in, inSize*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(iny_gpu, iny, inSize*sizeof(float2), hipMemcpyHostToDevice);

    int NUM_THREADS =BLOCK_SIZE;
    int NUM_BLOCKS = ((16640*4)+(NUM_THREADS-1)) / NUM_THREADS;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    lookuptable_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(in_gpu,iny_gpu,d_lookup, d_xxresult,d_yyresult,d_xyresult,d_yxresult,d_xxtemp,d_yytemp,d_xytemp,d_yxtemp);

    hipEventRecord(stop);

    // Synchronize to make sure all the streams have finished
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("%f\n",milliseconds);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    hipDeviceSynchronize();

    hipMemcpy(resultxx, d_xxresult, 16640 * sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(resultyy, d_yyresult, 16640 * sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(resultxy, d_xyresult, 16640 * sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(resultyx, d_yxresult, 16640 * sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(tempxx, d_xxtemp, temp_size* sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(tempyy, d_yytemp, temp_size* sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(tempxy, d_xytemp, temp_size* sizeof(float2), hipMemcpyDeviceToHost);
    
    hipFree(d_lookup);
    hipFree(d_xxresult);
    hipFree(d_yyresult);
    hipFree(d_xyresult);
    hipFree(d_yxresult);
    hipFree(in_gpu);
    hipFree(iny_gpu);
    hipFree(d_xxtemp);
    hipFree(d_yytemp);


    // for(int i=0;i<6;i++)
    // {
    //     printf("start->%f\n",tempxy[i].x);
    //     printf("end->%f\n",tempxy[i].y);
    // }

    float sum=0;
    for(int i=0;i<4;i++)
    {
        //printf("%f\n",tempyy[i].y);
        sum+=tempyy[i].y;
    }
    printf("Blocksum->%f\n",sum);

    sum=0;
    for(int i=0;i<1003;i++)
    {
        //printf("%f\n",tempyy[i].x);
        sum+=tempyy[i].x;
    }
    printf("Individualsum->%f\n",sum);
    // for(int i=0;i<16640;i++)
    // {
    //     printf("val->%f\n",tempyy[i].y);
    // }
    // for(int i=1;i<16640;i++)
    // {
    //     printf("i->%d\n",i);

    //     printf("EXP: %f\t RES: %f\n",exp[i].x,resultxx[i].x);
    //     assert(exp[i].x==resultxx[i].x);

    //     printf("EXP: %f\t RES: %f\n",exp[i].y,resultxx[i].y);
    //     assert(exp[i].y==resultxx[i].y);

    //     printf("EXP: %f\t RES: %f\n",exp[i].x,resultyy[i].x);
    //     assert(exp[i].x==resultyy[i].x);

    //     printf("EXP: %f\t RES: %f\n",exp[i].y,resultyy[i].y);
    //     assert(exp[i].y==resultyy[i].y);

    //     printf("EXP: %f\t RES: %f\n",exp[i].x,resultxy[i].x);
    //     assert(exp[i].x==resultxy[i].x);

    //     printf("EXP: %f\t RES: %f\n",exp[i].y,resultxy[i].y);
    //     assert(exp[i].y==resultxy[i].y);

    //     printf("EXP: %f\t RES: %f\n",exp[i].x,resultyx[i].x);
    //     assert(exp[i].x==resultyx[i].x);

    //     printf("EXP: %f\t RES: %f\n",exp[i].y,resultyx[i].y);
    //     assert(exp[i].y==resultyx[i].y);

    // }
    printf("test_cyclid_corr_accum passed\n");


    free(in);
    free(iny);
    free(exp);
    free(lookuptable);
    free(resultxx);
    free(resultyy);
    free(resultxy);
    free(resultyx);
    
    return 0;
}


int test_cyclid_corr_accum1() {
    cycfold_struct cs;

    // realistice size data set
    cs.ncyc = 128;
    cs.nlag = (cs.ncyc/2) + 1;
    cs.numTimeSamplesHfft = 256250;
    //int inSize2 = cs.numTimeSamplesHfft - cs.nlag - 1;
    cs.nBlocks = 1;
    cs.nchanPfb = 1;
    // init phase bins: not many phase bins, all samples use 0 but a few
    cs.numPhaseBins = 256;
    int phaseBinLookupSize = (2*cs.numTimeSamplesHfft) + cs.nlag - 2;

    unsigned *phaseBins;
    phaseBins = (unsigned *)malloc(phaseBinLookupSize *sizeof(unsigned));
    memset(phaseBins, 0, phaseBinLookupSize*sizeof(int));

    // spread out the phaseBins equally
    int phaseStep = phaseBinLookupSize / cs.numPhaseBins;
    //int phaseStepRem = phaseBinLookupSize % cs.numPhaseBins;
    for (int iphase=0; iphase<cs.numPhaseBins; iphase++) {
        int start = iphase*phaseStep;
        int end = (iphase+1)*phaseStep;
        for (int j=start; j<end; j++)
            phaseBins[j] = iphase;
    }

    // double check phase counts make sense
    int phaseCnts[cs.numPhaseBins];
    memset(phaseCnts, 0, cs.numPhaseBins*sizeof(int));
    for (int i = 0; i < cs.numTimeSamplesHfft; i++)
        for (int ilag = 0; ilag < cs.nlag; ilag++)
            phaseCnts[phaseBins[(2*i) + ilag]]++;

    // add them up
    int phaseCntTotal = 0;
    for (int i=0; i<cs.numPhaseBins; i++) {
        //printf("%d\n", phaseCnts[i]);
        phaseCntTotal += phaseCnts[i];
    }
    assert(phaseCntTotal == cs.numTimeSamplesHfft * cs.nlag);

    bool time = false;
    bool maxOccupancy = false;
    int rv = test_cyclid_corr_accum(&cs, phaseBins, maxOccupancy, time);
    free(phaseBins);
    return rv;
}

int test_cyclid_corr_accum2() {
    cycfold_struct cs;

    // very small data set
    cs.ncyc = 4;
    cs.nlag = (cs.ncyc/2) + 1;
    cs.numTimeSamplesHfft = 16;
    cs.nBlocks = 1;
    cs.nchanPfb = 1;
    // init phase bins: not many phase bins, all samples use 0
    cs.numPhaseBins = 4;

    int phaseBinLookupSize = (2*cs.numTimeSamplesHfft) + cs.nlag - 2;
    unsigned *phaseBins;
    phaseBins = (unsigned *)malloc(phaseBinLookupSize *sizeof(unsigned));
    memset(phaseBins, 0, phaseBinLookupSize*sizeof(int));

    bool time = false;
    bool maxOccupancy = false; // ignored for small data sets anyways
    int rv = test_cyclid_corr_accum(&cs, phaseBins, time, maxOccupancy);
    free(phaseBins);
    return rv;
}

int main() {
    printf("Lookup for all polarisation\n");
    fflush(stdout);
    test_cyclid_corr_accum1();
    //test_cyclid_corr_accum2();
}


