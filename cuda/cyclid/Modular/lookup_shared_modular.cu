/************************************************************************************
A modified code which computes lookuptable in the CPU and threads are launched in 2D
configuration where threadIdx.x are mapped to index in the output array and threads 
in y dimensions iterate a small portion of the 'for' loop and write its result in the 
shared memory. Zeroth thread in the x dimension add those values in the shared memory 
and writes it result in the output array. It uses modified input similar to real-world 
data where whole numbers are used and computes all polarisations.

Link: https://bitbucket.org/assessmentmcw/cyclid/src/master/lookup_shared_modular.cu

GPU Time: 14.5719 ms
CPU Time: 140.05 ms
***************************************************************************************/

#include <stdio.h>
#include <complex>
#include <assert.h>
#include<hip/hip_runtime.h>

struct cycfold_struct {
    unsigned ncyc;
    unsigned nlag;
    unsigned nchanPfb; 
    size_t numPhaseBins;
    unsigned numTimeSamplesHfft;
    unsigned nBlocks;
};


int GPU_BLOCK_SIZE = 256*4;

const int LOOP_SIZE=256;
const int BLOCK_SIZE=256;


inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void lookuptable_kernel(float2 *in1, float2 *in2, int2 *d_lookup, float2 *d_xxresult, float2 *d_yyresult, float2 *d_xyresult, float2 *d_yxresult) {
    
    int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int threadrow = threadIdx.y;
    if(rowIdx==1) printf("%d\n",d_lookup[rowIdx].x);
    __shared__ float2 memxx[BLOCK_SIZE][4];
    __shared__ float2 memyy[BLOCK_SIZE][4];
    __shared__ float2 memxy[BLOCK_SIZE][4];
    __shared__ float2 memyx[BLOCK_SIZE][4];

    if (rowIdx < 16640) {
        float2 sumxx,sumyy,sumxy,sumyx;
        sumxx.x = 0.0f;
        sumxx.y = 0.0f;
        sumyy.x = 0.0f;
        sumyy.y = 0.0f;
        sumxy.x = 0.0f;
        sumxy.y = 0.0f;
        sumyx.x = 0.0f;
        sumyx.y = 0.0f;

        int start=(threadrow*LOOP_SIZE)+1;
        int end=start+LOOP_SIZE-1;

        if(end>=1003) end=1002;
        
        for (int colIdx = start; colIdx <= end; colIdx++) {

            int2 current = d_lookup[rowIdx * 1003 + colIdx];

            if(current.x==-1) break;

            int current_x=current.x;
            int current_y=current.y;

            float in1y_x=in1[current_y].x;
            float in1x_x=in1[current_x].x;
            float in1y_y=in1[current_y].y;
            float in1x_y=in1[current_x].y;

            float in2y_x=in2[current_y].x;
            float in2x_x=in2[current_x].x;
            float in2y_y=in2[current_y].y;
            float in2x_y=in2[current_x].y;


            float2 product;

            //XX Corelation
            product.x = (in1y_x * in1x_x) + (in1y_y * in1x_y);
            product.y = (in1y_y * in1x_x)-(in1y_x *in1x_y);
            sumxx.x += product.x;      
            sumxx.y -= product.y;

            product.x = (in2y_x * in2x_x) + (in2y_y * in2x_y);
            product.y = (in2y_y * in2x_x)-(in2y_x *in2x_y);
            sumyy.x += product.x;      
            sumyy.y -= product.y;

            product.x = (in1y_x * in2x_x) + (in1y_y * in2x_y);
            product.y = (in1y_y * in2x_x)-(in1y_x*in2x_y);
            sumxy.x += product.x;      
            sumxy.y -= product.y;

            product.x = (in2y_x * in1x_x) + (in2y_y * in1x_y);
            product.y = (in2y_y * in1x_x)-(in2y_x *in1x_y);
            sumyx.x += product.x;      
            sumyx.y -= product.y;
            
        }

        //__syncthreads();

        memxx[threadIdx.x][threadIdx.y].x=sumxx.x;    
        memxx[threadIdx.x][threadIdx.y].y=sumxx.y;

        memyy[threadIdx.x][threadIdx.y].x=sumyy.x;    
        memyy[threadIdx.x][threadIdx.y].y=sumyy.y;

        memxy[threadIdx.x][threadIdx.y].x=sumxy.x;     
        memxy[threadIdx.x][threadIdx.y].y=sumxy.y;

        memyx[threadIdx.x][threadIdx.y].x=sumyx.x;     
        memyx[threadIdx.x][threadIdx.y].y=sumyx.y;
        
        __syncthreads();

        float2 xxfinal_sum,yyfinal_sum,xyfinal_sum,yxfinal_sum;
        xxfinal_sum.x=0;
        xxfinal_sum.y=0;
        yyfinal_sum.x=0;
        yyfinal_sum.y=0;
        xyfinal_sum.x=0;
        xyfinal_sum.y=0;
        yxfinal_sum.x=0;
        yxfinal_sum.y=0;

        if(threadIdx.y==0)
        {
            for(int i=0;i<4;i++)
            {
                xxfinal_sum.x+=memxx[threadIdx.x][i].x;
                xxfinal_sum.y+=memxx[threadIdx.x][i].y;
                yyfinal_sum.x+=memyy[threadIdx.x][i].x;
                yyfinal_sum.y+=memyy[threadIdx.x][i].y;
                xyfinal_sum.x+=memxy[threadIdx.x][i].x;
                xyfinal_sum.y+=memxy[threadIdx.x][i].y;
                yxfinal_sum.x+=memyx[threadIdx.x][i].x;
                yxfinal_sum.y+=memyx[threadIdx.x][i].y;
            }
            
            d_xxresult[rowIdx].x = xxfinal_sum.x;     
            d_xxresult[rowIdx].y = xxfinal_sum.y;

            d_yyresult[rowIdx].x = yyfinal_sum.x;     
            d_yyresult[rowIdx].y = yyfinal_sum.y;

            d_xyresult[rowIdx].x = xyfinal_sum.x;     
            d_xyresult[rowIdx].y = xyfinal_sum.y;

            d_yxresult[rowIdx].x = yxfinal_sum.x; 
            d_yxresult[rowIdx].y = yxfinal_sum.y;
        }
    
    }
}


void complexConjMult(float2 in1, float2 in2, float2 *tmp) {
    tmp->x = (in2.x * in1.x) - (in2.y * -1.0 * in1.y);
    tmp->y = (in2.x * -1.0 * in1.y) + (in2.y * in1.x);
    tmp->y = -1.0 * tmp->y;
}


void reference_code(float2 *in, float2 *exp, int inSize2, int nlag, int nchan, int ichan, int nPhaseBins,unsigned *phaseBins, bool verbose)
{
    int phaseBinIdx, phaseBin, expIdx;
    float2 tmp, in1, in2;
    for (int i = 0; i<inSize2; i++) {
        if (verbose)
            printf("\n%d\n", i);
        in1 = in[i];
        for (int ilag=0; ilag<nlag; ilag++) {
            if (verbose)
                printf(" ilag=%d ", ilag);
            in2 = in[i + ilag];
            complexConjMult(in1, in2, &tmp);

            // now accumulate in the right phase bin
            phaseBinIdx = (2*i)+ilag;
            phaseBin = phaseBins[phaseBinIdx];
            expIdx = (phaseBin * nlag * nchan) + (nlag * ichan) + ilag; 
            if (verbose) {
                printf(" pb=%d ",phaseBin);
                printf(" pi=%d ",phaseBinIdx);
                printf(" ei=%d ", expIdx);
            }
            // accumulate (fold)
            exp[expIdx].x += tmp.x;
            exp[expIdx].y += tmp.y;
        }
    
    }

    if (verbose)
        printf("\ncomputed expected results:\n");
    for (int iphase=0; iphase<nPhaseBins; iphase++) {
        if (verbose)
            printf("phase %d\n", iphase);
        for ( ichan=0; ichan<nchan; ichan++) {
            if (verbose)
                printf("chan %d\n", ichan);
            for (int ilag=0; ilag<nlag; ilag++ ) {
                expIdx = (iphase * nlag * nchan) + (nlag * ichan) + ilag; 
                if (verbose)
                    printf(" %f+%fi ", exp[expIdx].x, exp[expIdx].y);
            }
            if (verbose)
                printf("\n");
        }
    }
}


int call_all_polarisation_kernel(float2 *out,int inSize,int profileSize,int phaseBinLookupSize,int inSize2,int nlag,float2 *in,float2 *iny,unsigned* phaseBins,int nPhaseBins,int nchan,int iblock,int ichan,bool maxOccupancy,bool time,bool verbose,bool validate)
{
    printf("\n\nALL POLARISATION KERNEL\n\n");

    int phaseBinIdx, phaseBin, expIdx;

    clock_t start_time = clock();
    int2 *lookuptable;
    lookuptable= (int2 *)malloc(16640*1003* sizeof(int2));
    fflush(stdout);
    for(int c=0;c<16640*1003;c++)
    {
        lookuptable[c].x=-1;
    }
//     for(int i=0;i<1000;i++)
//   {
//     printf("%f\t%f\n",lookuptable[i].x,lookuptable[i].y);
//   }
    for (int i = 0; i<inSize2; i++) {
        for (int ilag=0; ilag<nlag; ilag++) {
            phaseBinIdx = (2*i)+ilag;
            phaseBin = phaseBins[phaseBinIdx];
            expIdx = (phaseBin * nlag * nchan) + (nlag * ichan) + ilag;
            int index =(lookuptable[expIdx*1003].x);
            if(index==-1)
            {
                lookuptable[expIdx*1003].x=1;
                lookuptable[expIdx*1003 + (lookuptable[expIdx*1003].x)].x = i;
                lookuptable[expIdx*1003 + (lookuptable[expIdx*1003].x)].y = i+ilag;
                lookuptable[expIdx*1003].x=(lookuptable[expIdx*1003].x)+1;
            }
            else
            {
                lookuptable[expIdx*1003 +(lookuptable[expIdx*1003].x)].x=i;
                lookuptable[expIdx*1003 + (lookuptable[expIdx*1003].x)].y=i+ilag;
                lookuptable[expIdx*1003].x= (lookuptable[expIdx*1003].x)+1;
            }
        }
    }
    
    
    clock_t end_time = clock();
    double elapsed_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
    //printf("%.6f\n", elapsed_time*1000);


    int2 *d_lookup;
    float2 *d_xxresult,*d_yyresult,*d_xyresult,*d_yxresult,*resultxx,*resultyy,*resultxy,*resultyx;
    
    
    resultxx= (float2 *)malloc(16640*sizeof(float2));
    resultyy= (float2 *)malloc(16640*sizeof(float2));
    resultxy= (float2 *)malloc(16640*sizeof(float2));
    resultyx= (float2 *)malloc(16640*sizeof(float2));

    for(int i=0;i<16640;i++)
    {
        resultxx[i].x=0.0f;
        resultxx[i].y=0.0f;
        resultyy[i].x=0.0f;
        resultyy[i].y=0.0f;
        resultxy[i].x=0.0f;
        resultxy[i].y=0.0f;
        resultyx[i].x=0.0f;
        resultyx[i].y=0.0f;
    }

    hipMalloc((void**)&d_lookup,16640*1003 * sizeof(int2));
    hipMalloc((void**)&d_xxresult, 16640* sizeof(float2));
    hipMalloc((void**)&d_yyresult, 16640* sizeof(float2));
    hipMalloc((void**)&d_xyresult, 16640* sizeof(float2));
    hipMalloc((void**)&d_yxresult, 16640* sizeof(float2));
    
    hipMemcpy(d_lookup,lookuptable, 16640*1003 * sizeof(int2), hipMemcpyHostToDevice);
    hipMemcpy(d_xxresult,resultxx, 16640* sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(d_yyresult,resultyy, 16640* sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(d_xyresult,resultxy, 16640* sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(d_yxresult,resultyx, 16640* sizeof(float2), hipMemcpyHostToDevice);


    float2 *in_gpu, *iny_gpu;
    hipMalloc((float2 **)&in_gpu, inSize*sizeof(float2));
    hipMalloc((float2 **)&iny_gpu, inSize*sizeof(float2));
    hipMemcpy(in_gpu, in, inSize*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(iny_gpu, iny, inSize*sizeof(float2), hipMemcpyHostToDevice);

    dim3 NUM_THREADS(BLOCK_SIZE,4);
    dim3 NUM_BLOCKS(((16640*4)/1024)+1,1,1);

    hipEvent_t startEvent, stopEvent;
    float ms;
    if (time) {
        checkCuda( hipEventCreate(&startEvent) );
        checkCuda( hipEventCreate(&stopEvent) );  
        checkCuda( hipEventRecord(startEvent, 0) );
    }

    lookuptable_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(in_gpu,iny_gpu,d_lookup, d_xxresult,d_yyresult,d_xyresult,d_yxresult);
    
    if (time) {
        checkCuda( hipEventRecord(stopEvent, 0) );
        checkCuda( hipEventSynchronize(stopEvent) );
        checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) ); 
        printf("%f\n", ms);
    }

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    hipDeviceSynchronize();

    hipMemcpy(resultxx, d_xxresult, 16640 * sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(resultyy, d_yyresult, 16640 * sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(resultxy, d_xyresult, 16640 * sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(resultyx, d_yxresult, 16640 * sizeof(float2), hipMemcpyDeviceToHost);
    
    hipFree(d_lookup);
    hipFree(d_xxresult);
    hipFree(d_yyresult);
    hipFree(d_xyresult);
    hipFree(d_yxresult);
    hipFree(in_gpu);
    hipFree(iny_gpu);

    if(validate)
    {   
        float2 tmp, in1, in2;
        float2 *exp;
        size_t profileSize = nPhaseBins * nchan * nlag;
        exp = (float2 *)malloc(profileSize * sizeof(float2));
        memset(exp, 0, profileSize*sizeof(float2));

        //Reference Code
        for (int i = 0; i<inSize2; i++) {
            in1 = in[i];
            for (int ilag=0; ilag<nlag; ilag++) {
                in2 = in[i + ilag];
                complexConjMult(in1, in2, &tmp);
                phaseBinIdx = (2*i)+ilag;
                phaseBin = phaseBins[phaseBinIdx];
                expIdx = (phaseBin * nlag * nchan) + (nlag * ichan) + ilag;
                exp[expIdx].x += tmp.x;
                exp[expIdx].y += tmp.y;
            }
        }

        int expIdx;
        float max_diffx=0;
        float max_diffy=0;
        if (verbose)
            printf("\nresults:\n");
        for (int iphase=0; iphase<nPhaseBins; iphase++) {
            if (verbose)
                printf("phase %d\n", iphase);
            for (int ichan=0; ichan<nchan; ichan++) {
                if (verbose)
                    printf("chan %d\n", ichan);
                for (int ilag=0; ilag<nlag; ilag++ ) {
                    expIdx = (iphase * nlag * nchan) + (nlag * ichan) + ilag; 
                    if (verbose)
                        printf(" %f+%fi ", resultxx[expIdx].x, resultxx[expIdx].y);
                    float diffx = abs(resultxx[expIdx].x - exp[expIdx].x);    
                    float diffy = abs(resultxx[expIdx].y - exp[expIdx].y);   
                    if(diffx>max_diffx)
                    {
                        max_diffx=diffx;
                    }
                    if(diffy>max_diffy)
                    {
                        max_diffy=diffy;
                    } 
                    float tol = 1e2;
                    if ((diffx > tol) || (diffy > tol)) {
                        printf("out[%d]=%f + %fi != exp[%d]=%f + %fi\n", expIdx, resultxx[expIdx].x, resultxx[expIdx].y, expIdx, exp[expIdx].x, exp[expIdx].y);
                        printf("diff x=%f y=%f\n", diffx, diffy);
                        return 1;
                    }
                }
                if (verbose)
                    printf("\n");
            }
        }
        printf("test_cyclid_corr_accum passed\n");
        printf("Max X:%f\tMax Y:%f\n",max_diffx,max_diffy);
    }

    return 1;
}
 
int main() {
    printf("Realworld_data_cyclid_gpu\n");

    cycfold_struct cs;

    cs.ncyc = 128;
    cs.nlag = (cs.ncyc/2) + 1; 
    cs.numTimeSamplesHfft = 256250;
    cs.nBlocks = 1;
    cs.nchanPfb = 1;
    cs.numPhaseBins = 256;

    int phaseBinLookupSize = (2*cs.numTimeSamplesHfft) + cs.nlag - 2;
    unsigned *phaseBins;
    phaseBins = (unsigned *)malloc(phaseBinLookupSize *sizeof(unsigned));
    memset(phaseBins, 0, phaseBinLookupSize*sizeof(int));

    int phaseStep = phaseBinLookupSize / cs.numPhaseBins;
    for (int iphase=0; iphase<cs.numPhaseBins; iphase++) {
        int start = iphase*phaseStep;
        int end = (iphase+1)*phaseStep;
        for (int j=start; j<end; j++)
            phaseBins[j] = iphase;
    }

    int phaseCnts[cs.numPhaseBins];
    memset(phaseCnts, 0, cs.numPhaseBins*sizeof(int));
    for (int i = 0; i < cs.numTimeSamplesHfft; i++)
        for (int ilag = 0; ilag < cs.nlag; ilag++)
            phaseCnts[phaseBins[(2*i) + ilag]]++;
    
    int phaseCntTotal = 0;
    for (int i=0; i<cs.numPhaseBins; i++) {
        phaseCntTotal += phaseCnts[i];
    }    
    assert(phaseCntTotal == cs.numTimeSamplesHfft * cs.nlag);    

    bool time = true;
    bool maxOccupancy = false;
    bool verbose = false;
    bool validate = true;

    int nlag = cs.nlag; 
    size_t inSize = cs.numTimeSamplesHfft;
    size_t inSize2 = inSize - nlag - 1;
    phaseBinLookupSize = (2*inSize) + nlag - 2;
    int nPhaseBins = cs.numPhaseBins;
    int nchan = cs.nchanPfb;
    int ichan = 0;
    int iblock = 0;
    
    float2 *in, *iny;
    in = (float2 *)malloc(inSize * sizeof(float2));
    iny = (float2 *)malloc(inSize * sizeof(float2));
    memset(in, 0, inSize*sizeof(float2));
    memset(iny, 0, inSize*sizeof(float2));
    int maxValue = 127; 
    int value = 0;
    float fvalue = 1.0;
    float imgDiv = 1.0;

    for (int i = 0; i<inSize; i++) {
        in[i].x = ((float)value) + fvalue;
        in[i].y = ( (float)((float)value)/imgDiv) + fvalue;
        iny[i].x = in[i].x;
        iny[i].y = in[i].y;
        value++;
        if (value + fvalue>maxValue)
            value=0;
    }
        
    if (verbose) {
        printf("inSize=%ld, inSize2=%ld\n", inSize, inSize2);
        printf("phaseBinLookupSize=%d\n", phaseBinLookupSize);
        for (int i = 0; i<inSize; i++) 
            printf("in[%d]=%f+%fi\n", i, in[i].x, in[i].y);
    }


    float2 *exp;
    size_t profileSize = nPhaseBins * nchan * nlag;
    exp = (float2 *)malloc(profileSize * sizeof(float2));
    memset(exp, 0, profileSize*sizeof(float2));

    reference_code(in,exp,inSize2,nlag,nchan,ichan,nPhaseBins,phaseBins,verbose);

    float2 *out;
    out = (float2 *)malloc(profileSize*sizeof(float2));

    call_all_polarisation_kernel(out,inSize,profileSize,phaseBinLookupSize,inSize2,nlag,in,iny,phaseBins,nPhaseBins,nchan,iblock,ichan,maxOccupancy,time,verbose,validate);

    free(phaseBins);
    free(in);
    free(iny);
    free(exp);
    free(out);

}