/****************************************************************************************
A modified code which uses shared memory to store the accumulating sum and then finally 
writing it into the global memory. It uses small input and computes only one polaristion.

Link: https://bitbucket.org/assessmentmcw/cyclid/src/master/shared_modular.cu

GPU Time: 0.123388ms
*****************************************************************************************/

#include <stdio.h>
#include <complex>
#include <assert.h>
#include<hip/hip_runtime.h>

struct cycfold_struct {
    unsigned ncyc;
    unsigned nlag;
    unsigned nchanPfb; 
    size_t numPhaseBins;
    unsigned numTimeSamplesHfft;
    unsigned nBlocks;
};


int GPU_BLOCK_SIZE = 256*4;

const int NUM_BINS=96;

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void cyclid_corr_accum_nlag_fast(float2 *in1, float2* in2, size_t size1, size_t size2, float2* out, int nlag, unsigned *phaseBins, int pfbChan, int numPfbChans, int iblock, int numPhaseBins, bool verbose) {
    // index into the data (in2)
    int inIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // index into the lagged data partially (in1)
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;
    // what phase bin does this result go into?
    unsigned phaseBin; 
    int lookupBlockLen = (size2*2) + nlag - 2;
    __shared__ float2 mem[NUM_BINS];
    for(int i=0;i<NUM_BINS;i++)
    {
        mem[i].x=0;
        mem[i].y=0;
    }
    __syncthreads();
    int phaseBinIdx = (iblock * lookupBlockLen) + (2*inIdx) + yIdx;
    //int phaseBinIdx = (iblock * lookupBlockLen) + (2*threadIdx.x) + threadIdx.y;
    phaseBin = phaseBins[phaseBinIdx];
    // and where in our output does the result go?
    int outIdx = (phaseBin * nlag * numPfbChans) + (nlag * pfbChan) + yIdx;
    //int outIdx = (phaseBin * nlag * numPfbChans) + (nlag * pfbChan) + threadIdx.y;
    size_t outSz = numPhaseBins * numPfbChans * nlag;
    float2 tmp;
    if ((inIdx<size2) && (outIdx<outSz) && (yIdx<nlag)) {
        // shift the in1 element, and mutliply by in2 conjugate element
        int j = inIdx+yIdx;

        tmp.x = (in1[j].x * in2[inIdx].x) - (in1[j].y * -1.0 * in2[inIdx].y);
        tmp.y = (in1[j].x * -1.0 * in2[inIdx].y) + (in1[j].y * in2[inIdx].x);
        // we will want to take the C2R FFT of the conjugate of this  
        tmp.y = -1.0 * tmp.y;

        atomicAdd(&mem[threadIdx.x*blockDim.x+threadIdx.y].x, tmp.x);
        atomicAdd(&mem[threadIdx.x*blockDim.x+threadIdx.y].y, tmp.y);
    }
    __syncthreads();

    if((inIdx<size2) && (outIdx<outSz) && (yIdx<nlag)){
        atomicAdd(&out[outIdx].x, mem[threadIdx.x*blockDim.x+threadIdx.y].x);
        atomicAdd(&out[outIdx].y, mem[threadIdx.x*blockDim.x+threadIdx.y].y);
    } 

}


void complexConjMult(float2 in1, float2 in2, float2 *tmp) {
    tmp->x = (in2.x * in1.x) - (in2.y * -1.0 * in1.y);
    tmp->y = (in2.x * -1.0 * in1.y) + (in2.y * in1.x);
    tmp->y = -1.0 * tmp->y;
}


void reference_code(float2 *in, float2 *exp, int inSize2, int nlag, int nchan, int ichan, int nPhaseBins,unsigned *phaseBins, bool verbose)
{
    int phaseBinIdx, phaseBin, expIdx;
    float2 tmp, in1, in2;
    for (int i = 0; i<inSize2; i++) {
        if (verbose)
            printf("\n%d\n", i);
        in1 = in[i];
        for (int ilag=0; ilag<nlag; ilag++) {
            if (verbose)
                printf(" ilag=%d ", ilag);
            in2 = in[i + ilag];
            complexConjMult(in1, in2, &tmp);

            // now accumulate in the right phase bin
            phaseBinIdx = (2*i)+ilag;
            phaseBin = phaseBins[phaseBinIdx];
            expIdx = (phaseBin * nlag * nchan) + (nlag * ichan) + ilag; 
            if (verbose) {
                printf(" pb=%d ",phaseBin);
                printf(" pi=%d ",phaseBinIdx);
                printf(" ei=%d ", expIdx);
            }
            // accumulate (fold)
            exp[expIdx].x += tmp.x;
            exp[expIdx].y += tmp.y;
        }
    
    }

    if (verbose)
        printf("\ncomputed expected results:\n");
    for (int iphase=0; iphase<nPhaseBins; iphase++) {
        if (verbose)
            printf("phase %d\n", iphase);
        for ( ichan=0; ichan<nchan; ichan++) {
            if (verbose)
                printf("chan %d\n", ichan);
            for (int ilag=0; ilag<nlag; ilag++ ) {
                expIdx = (iphase * nlag * nchan) + (nlag * ichan) + ilag; 
                if (verbose)
                    printf(" %f+%fi ", exp[expIdx].x, exp[expIdx].y);
            }
            if (verbose)
                printf("\n");
        }
    }
}


void call_fast_kernel(float2* out,int inSize,int profileSize,int phaseBinLookupSize,int inSize2,int nlag,float2 *in,float2 *iny,unsigned* phaseBins,int nPhaseBins,int nchan,int iblock,int ichan,bool maxOccupancy,bool time,bool verbose)
{
    printf("\n\nFAST KERNEL\n\n");
    float2 *in_gpu, *iny_gpu, *out_gpu, *outyy_gpu, *outxy_gpu, *outyx_gpu; 
    unsigned *phaseBins_gpu;
    hipMalloc((float2 **)&in_gpu, inSize*sizeof(float2));
    hipMalloc((float2 **)&iny_gpu, inSize*sizeof(float2));
    hipMalloc((float2 **)&out_gpu, profileSize*sizeof(float2));
    hipMalloc((float2 **)&outyy_gpu, profileSize*sizeof(float2));
    hipMalloc((float2 **)&outxy_gpu, profileSize*sizeof(float2));
    hipMalloc((float2 **)&outyx_gpu, profileSize*sizeof(float2));
    hipMalloc((unsigned **)&phaseBins_gpu, phaseBinLookupSize*sizeof(unsigned));
    hipMemset(out_gpu, 0, profileSize*sizeof(float2));
    hipMemset(outyy_gpu, 0, profileSize*sizeof(float2));
    hipMemset(outxy_gpu, 0, profileSize*sizeof(float2));
    hipMemset(outyx_gpu, 0, profileSize*sizeof(float2));

    hipMemcpy(in_gpu, in, inSize*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(iny_gpu, iny, inSize*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(phaseBins_gpu, phaseBins, phaseBinLookupSize*sizeof(unsigned), hipMemcpyHostToDevice);

    int gridX, gridY, threadX, threadY;
    if (inSize <  128) { 
        gridX = 1;
        gridY = 1;
        threadX = inSize2;
        threadY = nlag;
    } else {
        if (maxOccupancy) {
            int gpuGridSize = ((inSize2 + 256) / 256);
            gridX = gpuGridSize;
            gridY = (nlag+4)/4;
            threadX = 256; ///thisGpuBlockSize; //GPU_BLOCK_SIZE / nlag;
            threadY = 4; //nlag;
        } else {
            threadX = GPU_BLOCK_SIZE/nlag;
            threadY = nlag;
            gridX = (inSize2 + threadX) / threadX;
            gridY = 1;
        }

    }

    int numKernelCalls = gridX * gridY * threadX * threadY;

    if (verbose) {
        printf("inSize2=%d nlag=%d\n", inSize2, nlag);
        printf("grid x=%d, y=%d\n", gridX, gridY);
        printf("thread x=%d, y=%d\n", threadX, threadY);
        printf("num kernel calls: %d\n", numKernelCalls);
        printf("num needed: %d\n", inSize2 * nlag);
        float diffPct = ((numKernelCalls - (inSize2*nlag))/numKernelCalls)*100.0;
        printf("num null threads: %d, %f percent\n", numKernelCalls - (inSize2*nlag), diffPct);
    }

    dim3 grids(gridX, gridY, 1);
    dim3 threads(threadX, threadY, 1);
    ichan=0;

    assert(threadX * threadY <= GPU_BLOCK_SIZE);

    hipEvent_t startEvent, stopEvent;
    float ms;
    if (time) {
        checkCuda( hipEventCreate(&startEvent) );
        checkCuda( hipEventCreate(&stopEvent) );  
        checkCuda( hipEventRecord(startEvent, 0) );
    }

    cyclid_corr_accum_nlag_fast<<<grids, threads>>>(in_gpu, in_gpu, inSize,  inSize2, out_gpu, nlag, phaseBins_gpu, ichan, nchan, iblock, nPhaseBins, verbose);
    
    if (time) {
        checkCuda( hipEventRecord(stopEvent, 0) );
        checkCuda( hipEventSynchronize(stopEvent) );
        checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) ); 
        printf("%f\n", ms);
    }

    hipMemcpy(out, out_gpu, profileSize*sizeof(float2), hipMemcpyDeviceToHost);
    
    hipFree(in_gpu);
    hipFree(iny_gpu);
    hipFree(out_gpu);
    hipFree(outyy_gpu);
    hipFree(outxy_gpu);
    hipFree(outyx_gpu);
    hipFree(phaseBins_gpu);

}

int validate_results(int nPhaseBins,int nchan,int nlag,float2 *out,float2 *exp,bool verbose){
    int expIdx;
    if (verbose)
        printf("\nresults:\n");
    for (int iphase=0; iphase<nPhaseBins; iphase++) {
        if (verbose)
            printf("phase %d\n", iphase);
        for (int ichan=0; ichan<nchan; ichan++) {
            if (verbose)
                printf("chan %d\n", ichan);
            for (int ilag=0; ilag<nlag; ilag++ ) {
                expIdx = (iphase * nlag * nchan) + (nlag * ichan) + ilag; 
                if (verbose)
                    printf(" %f+%fi ", out[expIdx].x, out[expIdx].y);
                float diffx = abs(out[expIdx].x - exp[expIdx].x);    
                float diffy = abs(out[expIdx].y - exp[expIdx].y);    
                float tol = 1e2;
                if ((diffx > tol) || (diffy > tol)) {
                    printf("out[%d]=%f + %fi != exp[%d]=%f + %fi\n", expIdx, out[expIdx].x, out[expIdx].y, expIdx, exp[expIdx].x, exp[expIdx].y);
                    printf("diff x=%f y=%f\n", diffx, diffy);
                    return 1;
                }
            }
            if (verbose)
                printf("\n");
        }
    }
    printf("test_cyclid_corr_accum passed\n");
    return 1;
}
        

int main() {
    printf("Sample_data_cyclid_gpu\n");

    cycfold_struct cs;

    cs.ncyc = 4;
    cs.nlag = (cs.ncyc/2) + 1; 
    cs.numTimeSamplesHfft = 16;
    cs.nBlocks = 1;
    cs.nchanPfb = 1;
    cs.numPhaseBins = 4;

    int phaseBinLookupSize = (2*cs.numTimeSamplesHfft) + cs.nlag - 2;
    unsigned *phaseBins;
    phaseBins = (unsigned *)malloc(phaseBinLookupSize *sizeof(unsigned));
    memset(phaseBins, 0, phaseBinLookupSize*sizeof(int));

    bool time = true;
    bool maxOccupancy = false;
    bool verbose = false;

    int nlag = cs.nlag; 
    size_t inSize = cs.numTimeSamplesHfft;
    size_t inSize2 = inSize - nlag - 1;
    phaseBinLookupSize = (2*inSize) + nlag - 2;
    int nPhaseBins = cs.numPhaseBins;
    int nchan = cs.nchanPfb;
    int ichan = 0;
    int iblock = 0;
    
    float2 *in, *iny;
    in = (float2 *)malloc(inSize * sizeof(float2));
    iny = (float2 *)malloc(inSize * sizeof(float2));
    memset(in, 0, inSize*sizeof(float2));
    memset(iny, 0, inSize*sizeof(float2));
    int maxValue = 127; 
    int value = 0;
    float fvalue = 0.5;
    float imgDiv = 2.0;

    for (int i = 0; i<inSize; i++) {
        in[i].x = ((float)value) + fvalue;
        in[i].y = ( (float)((float)value)/imgDiv) + fvalue;
        iny[i].x = in[i].x;
        iny[i].y = in[i].y;
        value++;
        if (value + fvalue>maxValue)
            value=0;
    }
        
    if (verbose) {
        printf("inSize=%ld, inSize2=%ld\n", inSize, inSize2);
        printf("phaseBinLookupSize=%d\n", phaseBinLookupSize);
        for (int i = 0; i<inSize; i++) 
            printf("in[%d]=%f+%fi\n", i, in[i].x, in[i].y);
    }


    float2 *exp;
    size_t profileSize = nPhaseBins * nchan * nlag;
    exp = (float2 *)malloc(profileSize * sizeof(float2));
    memset(exp, 0, profileSize*sizeof(float2));

    reference_code(in,exp,inSize2,nlag,nchan,ichan,nPhaseBins,phaseBins,verbose);

    float2 *out;
    out = (float2 *)malloc(profileSize*sizeof(float2));

    call_fast_kernel(out,inSize,profileSize,phaseBinLookupSize,inSize2,nlag,in,iny,phaseBins,nPhaseBins,nchan,iblock,ichan,maxOccupancy,time,verbose);

    validate_results(nPhaseBins,nchan,nlag,out,exp,verbose);

    free(phaseBins);
    free(in);
    free(iny);
    free(exp);
    free(out);

}