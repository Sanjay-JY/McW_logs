#include "hip/hip_runtime.h"
#include <stdio.h>
#include <complex>
#include <assert.h>
#include<hip/hip_runtime.h>

struct cycfold_struct {
    unsigned ncyc;
    unsigned nlag;
    unsigned nchanPfb; 
    size_t numPhaseBins;
    unsigned numTimeSamplesHfft;
    unsigned nBlocks;
};


int GPU_BLOCK_SIZE = 256*4;


inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


__global__ void cyclid_corr_accum_nlag_fast(float2 *in1, float2* in2, size_t size1, size_t size2, float2* out, int nlag, unsigned *phaseBins, int pfbChan, int numPfbChans, int iblock, int numPhaseBins, bool verbose) {
    
    int inIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned phaseBin; 
    int lookupBlockLen = (size2*2) + nlag - 2;
    int phaseBinIdx = (iblock * lookupBlockLen) + (2*inIdx) + yIdx;
    phaseBin = phaseBins[phaseBinIdx];
    
    int outIdx = (phaseBin * nlag * numPfbChans) + (nlag * pfbChan) + yIdx;
    size_t outSz = numPhaseBins * numPfbChans * nlag;
    float2 tmp;

    if ((inIdx<size2) && (outIdx<outSz) && (yIdx<nlag)) {
        int j = inIdx+yIdx;
        tmp.x = (in1[j].x * in2[inIdx].x) - (in1[j].y * -1.0 * in2[inIdx].y);
        tmp.y = (in1[j].x * -1.0 * in2[inIdx].y) + (in1[j].y * in2[inIdx].x);  
        tmp.y = -1.0 * tmp.y;

        atomicAdd(&out[outIdx].x, tmp.x);
        atomicAdd(&out[outIdx].y, tmp.y);
    } 
}

<<<<<<< HEAD
__global__ void add_kernel(float2 *out,int2 *lookuptable)
{
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if(tid<16640)
    {
        for(int i=0;i<1003;i++)
        {
            out[tid].x+=lookuptable[(tid*1003)+i].x;
            out[tid].y+=lookuptable[(tid*1003)+i].y;

            out[tid].x+=lookuptable[(tid*1003)+i].x;
            out[tid].y+=lookuptable[(tid*1003)+i].y;

            out[tid].x+=lookuptable[(tid*1003)+i].x;
            out[tid].y+=lookuptable[(tid*1003)+i].y;

            out[tid].x+=lookuptable[(tid*1003)+i].x;
            out[tid].y+=lookuptable[(tid*1003)+i].y;
        }
    }
}

__global__ void cyclid_corr_accum_all_pols(int2 *lookuptable,float2 *in1, float2* in2, size_t size1, size_t size2, float2* outXX, float2* outYY, float2 *outXY, float2 *outYX, unsigned *phaseBins, int numPhaseBins, int numPfbChans, int nlag , int iblock, int pfbChan, size_t outSz, int lookupBlockLen, bool verbose, int maxvar) {
=======


__global__ void cyclid_corr_accum_all_pols(float2 *in1, float2* in2, size_t size1, size_t size2, float2* outXX, float2* outYY, float2 *outXY, float2 *outYX, unsigned *phaseBins, int numPhaseBins, int numPfbChans, int nlag , int iblock, int pfbChan, size_t outSz, int lookupBlockLen, bool verbose) {
>>>>>>> dbcd26cf1c03fa63e3b35a016d198ab11ca5e773

    int inIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned phaseBin ; 
    lookupBlockLen = (size2*2) + nlag - 2;
    int phaseBinIdx = (iblock * lookupBlockLen) + (2*inIdx) + yIdx;
    phaseBin = phaseBins[phaseBinIdx];

<<<<<<< HEAD
    int expIdx = (phaseBin * nlag * numPfbChans) + (nlag * pfbChan) + yIdx;
    int index;
    if(expIdx==0)
    {
        printf("%d\n",expIdx);
    }
    if ((inIdx<size2) && (expIdx<outSz) && (yIdx<nlag)) {
        int j = inIdx+yIdx;
        int i=inIdx;
        index = i%1003;
        // XX correlation
        lookuptable[expIdx*1003 + index].x=i;
        lookuptable[expIdx*1003 + index].y=j;       
=======
    int outIdx = (phaseBin * nlag * numPfbChans) + (nlag * pfbChan) + yIdx;
    float2 tmp;

    if ((inIdx<size2) && (outIdx<outSz) && (yIdx<nlag)) {
        int j = inIdx+yIdx;
        int i=inIdx;

                float in1i_x=in1[i].x;
                float in1i_y=in1[i].y;
                float in1j_x=in1[j].x;
                float in1j_y=in1[j].y;

                float in2i_x=in2[i].x;
                float in2i_y=in2[i].y;
                float in2j_x=in2[j].x;
                float in2j_y=in2[j].y;

                // XX correlation
                tmp.x = (in1j_x * in1i_x) + (in1j_y * in1i_y);   //+
                tmp.y = (in1j_y * in1i_x)-(in1j_x  * in1i_y);
              
               atomicAdd(&outXX[outIdx].x, tmp.x);
        atomicAdd(&outXX[outIdx].y, -tmp.y);

                // YY correlation
                tmp.x = (in2j_x * in2i_x) +(in2j_y * in2i_y);
                tmp.y = (in2j_y * in2i_x)-(in2j_x * in2i_y) ;
               
               atomicAdd(&outYY[outIdx].x, tmp.x);
        atomicAdd(&outYY[outIdx].y, -tmp.y);

                // XY correlation
                tmp.x = (in1j_x * in2i_x) +(in1j_y  * in2i_y);
                tmp.y = (in1j_y * in2i_x)-(in1j_x  * in2i_y)  ;
               
                atomicAdd(&outXY[outIdx].x, tmp.x);
        atomicAdd(&outXY[outIdx].y, -tmp.y);

 
                // YX correlation
                tmp.x = (in2j_x * in1i_x) +(in2j_y  * in1i_y);
                tmp.y = (in2j_y * in1i_x)-(in2j_x  * in1i_y)  ;
              

               atomicAdd(&outYX[outIdx].x, tmp.x);
        atomicAdd(&outYX[outIdx].y, -tmp.y);

        

>>>>>>> dbcd26cf1c03fa63e3b35a016d198ab11ca5e773
    } 
}


void complexConjMult(float2 in1, float2 in2, float2 *tmp) {
    tmp->x = (in2.x * in1.x) - (in2.y * -1.0 * in1.y);
    tmp->y = (in2.x * -1.0 * in1.y) + (in2.y * in1.x);
    tmp->y = -1.0 * tmp->y;
}


void reference_code(float2 *in, float2 *exp, int inSize2, int nlag, int nchan, int ichan, int nPhaseBins,unsigned *phaseBins, bool verbose)
{
    int phaseBinIdx, phaseBin, expIdx;
    float2 tmp, in1, in2;
    for (int i = 0; i<inSize2; i++) {
        if (verbose)
            printf("\n%d\n", i);
        in1 = in[i];
        for (int ilag=0; ilag<nlag; ilag++) {
            if (verbose)
                printf(" ilag=%d ", ilag);
            in2 = in[i + ilag];
            complexConjMult(in1, in2, &tmp);

            // now accumulate in the right phase bin
            phaseBinIdx = (2*i)+ilag;
            phaseBin = phaseBins[phaseBinIdx];
            expIdx = (phaseBin * nlag * nchan) + (nlag * ichan) + ilag; 
            if (verbose) {
                printf(" pb=%d ",phaseBin);
                printf(" pi=%d ",phaseBinIdx);
                printf(" ei=%d ", expIdx);
            }
            // accumulate (fold)
            exp[expIdx].x += tmp.x;
            exp[expIdx].y += tmp.y;
        }
    
    }

    if (verbose)
        printf("\ncomputed expected results:\n");
    for (int iphase=0; iphase<nPhaseBins; iphase++) {
        if (verbose)
            printf("phase %d\n", iphase);
        for ( ichan=0; ichan<nchan; ichan++) {
            if (verbose)
                printf("chan %d\n", ichan);
            for (int ilag=0; ilag<nlag; ilag++ ) {
                expIdx = (iphase * nlag * nchan) + (nlag * ichan) + ilag; 
                if (verbose)
                    printf(" %f+%fi ", exp[expIdx].x, exp[expIdx].y);
            }
            if (verbose)
                printf("\n");
        }
    }
}


<<<<<<< HEAD
void call_all_polarisation_kernel(float2 *out,int inSize,int profileSize,int phaseBinLookupSize,int inSize2,int nlag,float2 *in,float2 *iny,unsigned* phaseBins,int nPhaseBins,int nchan,int iblock,int ichan,bool maxOccupancy,bool time,bool verbose)
{
    int maxvar=0;
    int2 *lookuptable;
    lookuptable= (int2 *)malloc(16640*1003* sizeof(int2));
    fflush(stdout);
    for(int c=0;c<16640*1003;c++)
    {
        lookuptable[c].x=-1;
    }
    int2 *d_lookup;
    printf("\n\nALL POLARISATION KERNEL\n\n");
    float2 *in_gpu, *iny_gpu, *out_gpu, *outyy_gpu, *outxy_gpu, *outyx_gpu; 
    unsigned *phaseBins_gpu;
    hipMalloc((void**)&d_lookup,16640*1003*sizeof(int2));
    hipMemcpy(d_lookup,lookuptable, 16640*1003*sizeof(int2), hipMemcpyHostToDevice);
=======
void call_fast_kernel(float2* out,int inSize,int profileSize,int phaseBinLookupSize,int inSize2,int nlag,float2 *in,float2 *iny,unsigned* phaseBins,int nPhaseBins,int nchan,int iblock,int ichan,bool maxOccupancy,bool time,bool verbose)
{
    printf("\n\nFAST KERNEL\n\n");
    float2 *in_gpu, *iny_gpu, *out_gpu, *outyy_gpu, *outxy_gpu, *outyx_gpu; 
    unsigned *phaseBins_gpu;
>>>>>>> dbcd26cf1c03fa63e3b35a016d198ab11ca5e773
    hipMalloc((float2 **)&in_gpu, inSize*sizeof(float2));
    hipMalloc((float2 **)&iny_gpu, inSize*sizeof(float2));
    hipMalloc((float2 **)&out_gpu, profileSize*sizeof(float2));
    hipMalloc((float2 **)&outyy_gpu, profileSize*sizeof(float2));
    hipMalloc((float2 **)&outxy_gpu, profileSize*sizeof(float2));
    hipMalloc((float2 **)&outyx_gpu, profileSize*sizeof(float2));
    hipMalloc((unsigned **)&phaseBins_gpu, phaseBinLookupSize*sizeof(unsigned));
    hipMemset(out_gpu, 0, profileSize*sizeof(float2));
    hipMemset(outyy_gpu, 0, profileSize*sizeof(float2));
    hipMemset(outxy_gpu, 0, profileSize*sizeof(float2));
    hipMemset(outyx_gpu, 0, profileSize*sizeof(float2));

    hipMemcpy(in_gpu, in, inSize*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(iny_gpu, iny, inSize*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(phaseBins_gpu, phaseBins, phaseBinLookupSize*sizeof(unsigned), hipMemcpyHostToDevice);

    int gridX, gridY, threadX, threadY;
    if (inSize <  128) { 
        gridX = 1;
        gridY = 1;
        threadX = inSize2;
        threadY = nlag;
    } else {
        if (maxOccupancy) {
            int gpuGridSize = ((inSize2 + 256) / 256);
            gridX = gpuGridSize;
            gridY = (nlag+4)/4;
            threadX = 256; ///thisGpuBlockSize; //GPU_BLOCK_SIZE / nlag;
            threadY = 4; //nlag;
        } else {
            threadX = GPU_BLOCK_SIZE/nlag;
            threadY = nlag;
            gridX = (inSize2 + threadX) / threadX;
            gridY = 1;
        }

    }

    int numKernelCalls = gridX * gridY * threadX * threadY;

    if (verbose) {
        printf("inSize2=%d nlag=%d\n", inSize2, nlag);
        printf("grid x=%d, y=%d\n", gridX, gridY);
        printf("thread x=%d, y=%d\n", threadX, threadY);
        printf("num kernel calls: %d\n", numKernelCalls);
        printf("num needed: %d\n", inSize2 * nlag);
        float diffPct = ((numKernelCalls - (inSize2*nlag))/numKernelCalls)*100.0;
        printf("num null threads: %d, %f percent\n", numKernelCalls - (inSize2*nlag), diffPct);
    }

    dim3 grids(gridX, gridY, 1);
    dim3 threads(threadX, threadY, 1);
    ichan=0;

    assert(threadX * threadY <= GPU_BLOCK_SIZE);

    hipEvent_t startEvent, stopEvent;
    float ms;
    if (time) {
        checkCuda( hipEventCreate(&startEvent) );
        checkCuda( hipEventCreate(&stopEvent) );  
        checkCuda( hipEventRecord(startEvent, 0) );
    }

<<<<<<< HEAD
    cyclid_corr_accum_all_pols<<<grids,threads>>>(d_lookup,in_gpu, iny_gpu, inSize, inSize2, out_gpu, outyy_gpu, outxy_gpu, outyx_gpu, phaseBins_gpu, nPhaseBins, nchan, nlag, iblock, ichan, profileSize, phaseBinLookupSize, verbose, maxvar);
    
    // dim3 threadsa(1024, 1, 1);
    // dim3 gridsa((16640/1024)+1, 1, 1);

    //add_kernel<<<gridsa,threadsa>>>(out_gpu,d_lookup);
=======
    cyclid_corr_accum_nlag_fast<<<grids, threads>>>(in_gpu, in_gpu, inSize,  inSize2, out_gpu, nlag, phaseBins_gpu, ichan, nchan, iblock, nPhaseBins, verbose);
    
    if (time) {
        checkCuda( hipEventRecord(stopEvent, 0) );
        checkCuda( hipEventSynchronize(stopEvent) );
        checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) ); 
        printf("%f\n", ms);
    }

    hipMemcpy(out, out_gpu, profileSize*sizeof(float2), hipMemcpyDeviceToHost);

    hipFree(in_gpu);
    hipFree(iny_gpu);
    hipFree(out_gpu);
    hipFree(outyy_gpu);
    hipFree(outxy_gpu);
    hipFree(outyx_gpu);
    hipFree(phaseBins_gpu);

}

void call_all_polarisation_kernel(float2 *out,int inSize,int profileSize,int phaseBinLookupSize,int inSize2,int nlag,float2 *in,float2 *iny,unsigned* phaseBins,int nPhaseBins,int nchan,int iblock,int ichan,bool maxOccupancy,bool time,bool verbose)
{
    printf("\n\nALL POLARISATION KERNEL\n\n");
    float2 *in_gpu, *iny_gpu, *out_gpu, *outyy_gpu, *outxy_gpu, *outyx_gpu; 
    unsigned *phaseBins_gpu;
    hipMalloc((float2 **)&in_gpu, inSize*sizeof(float2));
    hipMalloc((float2 **)&iny_gpu, inSize*sizeof(float2));
    hipMalloc((float2 **)&out_gpu, profileSize*sizeof(float2));
    hipMalloc((float2 **)&outyy_gpu, profileSize*sizeof(float2));
    hipMalloc((float2 **)&outxy_gpu, profileSize*sizeof(float2));
    hipMalloc((float2 **)&outyx_gpu, profileSize*sizeof(float2));
    hipMalloc((unsigned **)&phaseBins_gpu, phaseBinLookupSize*sizeof(unsigned));
    hipMemset(out_gpu, 0, profileSize*sizeof(float2));
    hipMemset(outyy_gpu, 0, profileSize*sizeof(float2));
    hipMemset(outxy_gpu, 0, profileSize*sizeof(float2));
    hipMemset(outyx_gpu, 0, profileSize*sizeof(float2));

    hipMemcpy(in_gpu, in, inSize*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(iny_gpu, iny, inSize*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(phaseBins_gpu, phaseBins, phaseBinLookupSize*sizeof(unsigned), hipMemcpyHostToDevice);

    int gridX, gridY, threadX, threadY;
    if (inSize <  128) { 
        gridX = 1;
        gridY = 1;
        threadX = inSize2;
        threadY = nlag;
    } else {
        if (maxOccupancy) {
            int gpuGridSize = ((inSize2 + 256) / 256);
            gridX = gpuGridSize;
            gridY = (nlag+4)/4;
            threadX = 256; ///thisGpuBlockSize; //GPU_BLOCK_SIZE / nlag;
            threadY = 4; //nlag;
        } else {
            threadX = GPU_BLOCK_SIZE/nlag;
            threadY = nlag;
            gridX = (inSize2 + threadX) / threadX;
            gridY = 1;
        }

    }

    int numKernelCalls = gridX * gridY * threadX * threadY;

    if (verbose) {
        printf("inSize2=%d nlag=%d\n", inSize2, nlag);
        printf("grid x=%d, y=%d\n", gridX, gridY);
        printf("thread x=%d, y=%d\n", threadX, threadY);
        printf("num kernel calls: %d\n", numKernelCalls);
        printf("num needed: %d\n", inSize2 * nlag);
        float diffPct = ((numKernelCalls - (inSize2*nlag))/numKernelCalls)*100.0;
        printf("num null threads: %d, %f percent\n", numKernelCalls - (inSize2*nlag), diffPct);
    }

    dim3 grids(gridX, gridY, 1);
    dim3 threads(threadX, threadY, 1);
    ichan=0;

    assert(threadX * threadY <= GPU_BLOCK_SIZE);

    hipEvent_t startEvent, stopEvent;
    float ms;
    if (time) {
        checkCuda( hipEventCreate(&startEvent) );
        checkCuda( hipEventCreate(&stopEvent) );  
        checkCuda( hipEventRecord(startEvent, 0) );
    }

    cyclid_corr_accum_all_pols<<<grids,threads>>>(in_gpu, iny_gpu, inSize, inSize2, out_gpu, outyy_gpu, outxy_gpu, outyx_gpu, phaseBins_gpu, nPhaseBins, nchan, nlag, iblock, ichan, profileSize, phaseBinLookupSize, verbose);
    
>>>>>>> dbcd26cf1c03fa63e3b35a016d198ab11ca5e773
    if (time) {
        checkCuda( hipEventRecord(stopEvent, 0) );
        checkCuda( hipEventSynchronize(stopEvent) );
        checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) ); 
        printf("%f\n", ms);
    }

    hipMemcpy(out, out_gpu, profileSize*sizeof(float2), hipMemcpyDeviceToHost);
<<<<<<< HEAD
    printf("Max: %d\n",maxvar);
=======
>>>>>>> dbcd26cf1c03fa63e3b35a016d198ab11ca5e773

    hipFree(in_gpu);
    hipFree(iny_gpu);
    hipFree(out_gpu);
    hipFree(outyy_gpu);
    hipFree(outxy_gpu);
    hipFree(outyx_gpu);
    hipFree(phaseBins_gpu);

}


int validate_results(int nPhaseBins,int nchan,int nlag,float2 *out,float2 *exp,bool verbose){
    int expIdx;
    if (verbose)
        printf("\nresults:\n");
    for (int iphase=0; iphase<nPhaseBins; iphase++) {
        if (verbose)
            printf("phase %d\n", iphase);
        for (int ichan=0; ichan<nchan; ichan++) {
            if (verbose)
                printf("chan %d\n", ichan);
            for (int ilag=0; ilag<nlag; ilag++ ) {
                expIdx = (iphase * nlag * nchan) + (nlag * ichan) + ilag; 
                if (verbose)
                    printf(" %f+%fi ", out[expIdx].x, out[expIdx].y);
                float diffx = abs(out[expIdx].x - exp[expIdx].x);    
                float diffy = abs(out[expIdx].y - exp[expIdx].y);    
                float tol = 1e2;
                if ((diffx > tol) || (diffy > tol)) {
                    printf("out[%d]=%f + %fi != exp[%d]=%f + %fi\n", expIdx, out[expIdx].x, out[expIdx].y, expIdx, exp[expIdx].x, exp[expIdx].y);
                    printf("diff x=%f y=%f\n", diffx, diffy);
                    return 1;
                }
            }
            if (verbose)
                printf("\n");
        }
    }
    printf("test_cyclid_corr_accum passed\n");
    return 1;
}
    
    

int main() {
    printf("Realworld_data_cyclid_gpu\n");

    cycfold_struct cs;

    cs.ncyc = 128;
    cs.nlag = (cs.ncyc/2) + 1; 
    cs.numTimeSamplesHfft = 256250;
    cs.nBlocks = 1;
    cs.nchanPfb = 1;
    cs.numPhaseBins = 256;

    int phaseBinLookupSize = (2*cs.numTimeSamplesHfft) + cs.nlag - 2;
    unsigned *phaseBins;
    phaseBins = (unsigned *)malloc(phaseBinLookupSize *sizeof(unsigned));
    memset(phaseBins, 0, phaseBinLookupSize*sizeof(int));

    int phaseStep = phaseBinLookupSize / cs.numPhaseBins;
    for (int iphase=0; iphase<cs.numPhaseBins; iphase++) {
        int start = iphase*phaseStep;
        int end = (iphase+1)*phaseStep;
        for (int j=start; j<end; j++)
            phaseBins[j] = iphase;
    }

    int phaseCnts[cs.numPhaseBins];
    memset(phaseCnts, 0, cs.numPhaseBins*sizeof(int));
    for (int i = 0; i < cs.numTimeSamplesHfft; i++)
        for (int ilag = 0; ilag < cs.nlag; ilag++)
            phaseCnts[phaseBins[(2*i) + ilag]]++;
    
    int phaseCntTotal = 0;
    for (int i=0; i<cs.numPhaseBins; i++) {
        phaseCntTotal += phaseCnts[i];
    }    
    assert(phaseCntTotal == cs.numTimeSamplesHfft * cs.nlag);    

    bool time = true;
    bool maxOccupancy = false;
    bool verbose = false;

    int nlag = cs.nlag; 
    size_t inSize = cs.numTimeSamplesHfft;
    size_t inSize2 = inSize - nlag - 1;
    phaseBinLookupSize = (2*inSize) + nlag - 2;
    int nPhaseBins = cs.numPhaseBins;
    int nchan = cs.nchanPfb;
    int ichan = 0;
    int iblock = 0;
    
    float2 *in, *iny;
    in = (float2 *)malloc(inSize * sizeof(float2));
    iny = (float2 *)malloc(inSize * sizeof(float2));
    memset(in, 0, inSize*sizeof(float2));
    memset(iny, 0, inSize*sizeof(float2));
    int maxValue = 127; 
    int value = 0;
    float fvalue = 0.5;
    float imgDiv = 2.0;

    for (int i = 0; i<inSize; i++) {
        in[i].x = ((float)value) + fvalue;
        in[i].y = ( (float)((float)value)/imgDiv) + fvalue;
        iny[i].x = in[i].x;
        iny[i].y = in[i].y;
        value++;
        if (value + fvalue>maxValue)
            value=0;
    }
        
    if (verbose) {
        printf("inSize=%ld, inSize2=%ld\n", inSize, inSize2);
        printf("phaseBinLookupSize=%d\n", phaseBinLookupSize);
        for (int i = 0; i<inSize; i++) 
            printf("in[%d]=%f+%fi\n", i, in[i].x, in[i].y);
    }


    float2 *exp;
    size_t profileSize = nPhaseBins * nchan * nlag;
    exp = (float2 *)malloc(profileSize * sizeof(float2));
    memset(exp, 0, profileSize*sizeof(float2));

    reference_code(in,exp,inSize2,nlag,nchan,ichan,nPhaseBins,phaseBins,verbose);

    float2 *out;
    out = (float2 *)malloc(profileSize*sizeof(float2));

    //call_fast_kernel(out,inSize,profileSize,phaseBinLookupSize,inSize2,nlag,in,iny,phaseBins,nPhaseBins,nchan,iblock,ichan,maxOccupancy,time,verbose);
    call_all_polarisation_kernel(out,inSize,profileSize,phaseBinLookupSize,inSize2,nlag,in,iny,phaseBins,nPhaseBins,nchan,iblock,ichan,maxOccupancy,time,verbose);

    validate_results(nPhaseBins,nchan,nlag,out,exp,verbose);

    free(phaseBins);
    free(in);
    free(iny);
    free(exp);
    free(out);

}